#include <cstdio>
#include <cstdlib>
#include <math.h>
#include <hip/hip_runtime.h>
#include <chrono>
#include <fstream>

#include <iostream>
using namespace std::chrono;
clock_t start = clock();

#define INF 200

//RJ: defining the variables for blocked implementation
int n;
int blocking_factor = 25;
int rounds = 0;

__global__ void gpu_submatrix_fw_krnl_p1(int row_offset, int colm_offset, int *arr, int rnd_id, int blocking_factor, int n){
    int w = 0;
    //int b_k =  threadIdx.x;
    //printf("Received thread idx = %d\n", b_k);
    //printf("Print order: i_j_idx, i_k_idx, k_j_idx \n");
    for(int b_k = 0; b_k < blocking_factor; b_k++){
            //accessing each element in the submatrix
            for(int b_i = 0; b_i < blocking_factor; b_i++){
                for(int b_j = 0; b_j < blocking_factor; b_j++){
                    //index for i,j
                    int i_j_idx = (row_offset + b_i) * n + (colm_offset * blocking_factor) + b_j; //x_posn + y_posn
                    int i_k_idx = (row_offset + b_i) * n + (colm_offset * blocking_factor) + b_k; //x_posn + y_posn
                    int k_j_idx = (row_offset + b_k) * n + (colm_offset * blocking_factor) + b_j; //x_posn + y_posn
                    if((w = arr[i_k_idx] + arr[k_j_idx]) < arr[i_j_idx]){
                        arr[i_j_idx] = w;
                    }
                    //printf("%d \t %d\t %d\n", arr[i_j_idx], arr[i_k_idx], arr[k_j_idx]);
                    //printf("%d\t%d\t%d\n", i_j_idx, i_k_idx, k_j_idx);
                }
                //printf("\n");    
            }
        }
}


__global__ void gpu_submatrix_fw_krnl_p2(int row_offset, int colm_offset, int *arr, int rnd_id, int blocking_factor, int n){
    int w = 0;
    colm_offset = threadIdx.x;
    for(int b_k = 0; b_k < blocking_factor; b_k++){
            //accessing each element in the submatrix
            for(int b_i = 0; b_i < blocking_factor; b_i++){
                for(int b_j = 0; b_j < blocking_factor; b_j++){
                    //index for i,j
                    int i_j_idx = (row_offset + b_i) * n + (colm_offset * blocking_factor) + b_j; //x_posn + y_posn
                    int i_k_idx = (rnd_id * blocking_factor + b_i) * n + (rnd_id * blocking_factor) + b_k; //x_posn + y_posn
                    int k_j_idx = (rnd_id * blocking_factor + b_k) * n + (colm_offset * blocking_factor) + b_j; //x_posn + y_posn
                    if((w = arr[i_k_idx] + arr[k_j_idx]) < arr[i_j_idx]){
                        arr[i_j_idx] = w;
                    }
                    //printf("%d\t%d\t%d\n", i_j_idx, i_k_idx, k_j_idx);
                    //printf("%d \t %d\t %d\n", arr[i_j_idx], arr[i_k_idx], arr[k_j_idx]);
                }
                //printf("\n");    
            }
        }
}

__global__ void gpu_submatrix_fw_krnl_p3(int row_offset, int colm_offset, int *arr, int rnd_id, int blocking_factor, int n){
    int w = 0;
    row_offset = threadIdx.x * blocking_factor;
    for(int b_k = 0; b_k < blocking_factor; b_k++){
            //accessing each element in the submatrix
            for(int b_i = 0; b_i < blocking_factor; b_i++){
                for(int b_j = 0; b_j < blocking_factor; b_j++){
                    //index for i,j
                    int i_j_idx = (row_offset + b_i) * n + (colm_offset * blocking_factor) + b_j; //x_posn + y_posn
                    int i_k_idx = (row_offset + b_i) * n + (rnd_id * blocking_factor) + b_k; //x_posn + y_posn
                    int k_j_idx = (rnd_id * blocking_factor + b_k) * n + (colm_offset * blocking_factor) + b_j; //x_posn + y_posn
                    if((w = arr[i_k_idx] + arr[k_j_idx]) < arr[i_j_idx]){
                        arr[i_j_idx] = w;
                    }
                    //printf("%d\t%d\t%d\n", i_j_idx, i_k_idx, k_j_idx);
                    //printf("%d \t %d\t %d\n", arr[i_j_idx], arr[i_k_idx], arr[k_j_idx]);
                }
                //printf("\n");    
            }
        }
}

__global__ void gpu_submatrix_fw_krnl_p4(int row_offset, int colm_offset, int *arr, int rnd_id, int blocking_factor, int n){
    int w = 0;
    row_offset = threadIdx.x * blocking_factor;
    colm_offset = blockIdx.x;
    //printf("(x,y) is: (%d, %d)\n", threadIdx.x, blockIdx.x);
    for(int b_k = 0; b_k < blocking_factor; b_k++){
            //accessing each element in the submatrix
            for(int b_i = 0; b_i < blocking_factor; b_i++){
                for(int b_j = 0; b_j < blocking_factor; b_j++){
                    //index for i,j
                    int i_j_idx = (row_offset + b_i) * n + (colm_offset * blocking_factor) + b_j; //x_posn + y_posn
                    int i_k_idx = (row_offset + b_i) * n + (rnd_id * blocking_factor) + b_k; //x_posn + y_posn
                    int k_j_idx = (rnd_id * blocking_factor + b_k) * n + (colm_offset * blocking_factor) + b_j; //x_posn + y_posn
                    if((w = arr[i_k_idx] + arr[k_j_idx]) < arr[i_j_idx]){
                        arr[i_j_idx] = w;
                    }
                    //printf("%d \t %d\t %d\n", arr[i_j_idx], arr[i_k_idx], arr[k_j_idx]);
                    //printf("%d\t%d\t%d\n", i_j_idx, i_k_idx, k_j_idx);
                }
                //printf("\n");    
            }
        }
}


int main(int argc, char** argv) {
    int m, *d;
    // input
    FILE *infile = fopen(argv[1], "r");
    fscanf(infile, "%d %d", &n, &m);
    rounds = ceil(n/(double)blocking_factor);
    d = (int *) malloc(sizeof(int *) * n * n);
    for (int i = 0; i < n * n; ++i) d[i] = INF;
    int a, b, w;
    
    for (int i = 0; i < m; ++i) {
        fscanf(infile, "%d %d %d", &a, &b, &w);
        d[a * n + b] = d[b * n + a] = w;
    }
    fclose(infile);

    
    
    auto start = high_resolution_clock::now();
    //RJ: GPU init
    int *gpu_d;
    hipMalloc((void **)&gpu_d, sizeof(int *) * n * n);
    hipMemcpy(gpu_d, d, sizeof(int)* n * n, hipMemcpyHostToDevice);
    dim3 block_dim(blocking_factor,blocking_factor);
    //RJ: notes
    // N/B rounds --> 3 phases --> B iterations
    // seems like phase2, phase 3 can be parallelized.
    
    //auto stop = high_resolution_clock::now();
    //auto duration = duration_cast<microseconds>(stop - stop);
    printf("Rounds are: %d\n", rounds);
    for (int rnd_id = 0; rnd_id < rounds; rnd_id++){ // number of rounds
        //do 3 phases in each round
        //Phase1 -- computing on the pivot block
        //(1) find which cell to work on. (2) apply blocked FW

        int cell_rowid = rnd_id * blocking_factor; // row_id
        int cell_colmid = rnd_id * blocking_factor; // row_id
        //GPU declaration and kernel launch:
        dim3 dimGrid = (1);
        dim3 dimBlock = 1;
        //start = high_resolution_clock::now();
        gpu_submatrix_fw_krnl_p1<<<dimGrid, dimBlock>>>(cell_rowid, rnd_id, gpu_d, rnd_id, blocking_factor, n);
        hipDeviceSynchronize();
        //stop = high_resolution_clock::now();
        //duration += duration_cast<microseconds>(stop - start);
        // cudaMemcpy(d, gpu_d, sizeof(int)* n * n, cudaMemcpyDeviceToHost);
        // cudaDeviceSynchronize();
        // cudaMemcpy(gpu_d, d, sizeof(int)* n * n, cudaMemcpyHostToDevice);
        // cudaDeviceSynchronize();
        //Phase2 -- apply fw on the pivot-row and pivot-colm
        // (1) cover row
        // (2) cover colm
        //pivot block is (rnd_id, rnd_id)
        //par1: cover row
        
        dimGrid = (1);
        dimBlock = rounds;

        //start = high_resolution_clock::now();
        gpu_submatrix_fw_krnl_p2<<<dimGrid, dimBlock>>>(cell_rowid, 0, gpu_d, rnd_id, blocking_factor, n);
        // for(int block = 0; block < rounds; block++){
        //    if(block != rnd_id){
        //     gpu_submatrix_fw_krnl_p2<<<dimGrid, dimBlock, blocking_factor * blocking_factor * sizeof(int)*2>>>(cell_rowid, block, gpu_d, rnd_id, blocking_factor, n);
        //    }
        // }

        //par2: cover colm
        gpu_submatrix_fw_krnl_p3<<<dimGrid, dimBlock>>>(0*blocking_factor, rnd_id, gpu_d, rnd_id, blocking_factor, n);
        // for(int block = 0; block < rounds; block++){
        //    if(block != rnd_id){
        //     gpu_submatrix_fw_krnl_p3<<<dimGrid, dimBlock, blocking_factor * blocking_factor * sizeof(int)*2>>>(block*blocking_factor, rnd_id, gpu_d, rnd_id, blocking_factor, n);
        //    }
        // }

        // cudaMemcpy(d, gpu_d, sizeof(int)* n * n, cudaMemcpyDeviceToHost);
        // cudaDeviceSynchronize();
        // cudaMemcpy(gpu_d, d, sizeof(int)* n * n, cudaMemcpyHostToDevice);
        hipDeviceSynchronize();
        

        //Phase3 -- apply fw on the non-pivot blocks
        dimGrid = (rounds);
        dimBlock = (rounds);
        gpu_submatrix_fw_krnl_p4<<<dimGrid, dimBlock>>>(0*blocking_factor, 0, gpu_d, rnd_id, blocking_factor, n);
        // for(int rnd_i = 0; rnd_i < rounds; rnd_i++){
        //     if(rnd_i != rnd_id){
        //         for(int rnd_j = 0; rnd_j < rounds; rnd_j++){
        //             if(rnd_j != rnd_id){
        //                 gpu_submatrix_fw_krnl_p4<<<dimGrid, dimBlock>>>(rnd_i*blocking_factor, rnd_j, gpu_d, rnd_id, blocking_factor, n);
        //                 //submatrix_fw_krnl_p4(rnd_i*blocking_factor, rnd_j, d, rnd_id);
        //             }
        //         }
        //     }
        // }

        // cudaMemcpy(d, gpu_d, sizeof(int)* n * n, cudaMemcpyDeviceToHost);
        // cudaDeviceSynchronize();
        // cudaMemcpy(gpu_d, d, sizeof(int)* n * n, cudaMemcpyHostToDevice);
        hipDeviceSynchronize();
        // stop = high_resolution_clock::now();
        // duration += duration_cast<microseconds>(stop - start);
        
    }
    
    //Obtain back the data from GPU
    hipMemcpy(d, gpu_d, sizeof(int)* n * n, hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    auto stop = high_resolution_clock::now();
    auto duration = duration_cast<microseconds>(stop - start);
    std::cout << ">>>> Time to compute on GPU: " << duration.count()/(1000000.0) << std::endl;
    
    // ouput
    FILE *outfile = fopen(argv[2], "w");
    for (int i = 0; i < n; ++i) {
        for (int j = 0; j < n; ++j) {
            fprintf(outfile, "%d%s",
                (i == j ? 0 : d[i * n + j]),
                (j == n - 1 ? " \n" : " ")
            );
        }
    }
    free(d);
}