#include <cstdio>
#include <cstdlib>
#include <math.h>
#include <hip/hip_runtime.h>
#include <chrono>
#include <fstream>

#include <iostream>
using namespace std::chrono;

#define INF 200

//RJ: defining the variables for blocked implementation
int n;
int blocking_factor = 25;
int rounds = 0;

__global__ void gpu_submatrix_fw_krnl_p1(int row_offset, int colm_offset, int *arr, int rnd_id, int blocking_factor, int n){
    int w = 0;
    for(int b_k = 0; b_k < blocking_factor; b_k++){
            //accessing each element in the submatrix
            for(int b_i = 0; b_i < blocking_factor; b_i++){
                for(int b_j = 0; b_j < blocking_factor; b_j++){
                    //index for i,j
                    int i_j_idx = (row_offset + b_i) * n + (colm_offset * blocking_factor) + b_j; //x_posn + y_posn
                    int i_k_idx = (row_offset + b_i) * n + (colm_offset * blocking_factor) + b_k; //x_posn + y_posn
                    int k_j_idx = (row_offset + b_k) * n + (colm_offset * blocking_factor) + b_j; //x_posn + y_posn
                    if((w = arr[i_k_idx] + arr[k_j_idx]) < arr[i_j_idx]){
                        arr[i_j_idx] = w;
                    }
                }
            }
        }
}


__global__ void gpu_submatrix_fw_krnl_p2(int row_offset, int colm_offset, int *arr, int rnd_id, int blocking_factor, int n){
    int w = 0;
    colm_offset = threadIdx.x;
    for(int b_k = 0; b_k < blocking_factor; b_k++){
            //accessing each element in the submatrix
            for(int b_i = 0; b_i < blocking_factor; b_i++){
                for(int b_j = 0; b_j < blocking_factor; b_j++){
                    //index for i,j
                    int i_j_idx = (row_offset + b_i) * n + (colm_offset * blocking_factor) + b_j; //x_posn + y_posn
                    int i_k_idx = (rnd_id * blocking_factor + b_i) * n + (rnd_id * blocking_factor) + b_k; //x_posn + y_posn
                    int k_j_idx = (rnd_id * blocking_factor + b_k) * n + (colm_offset * blocking_factor) + b_j; //x_posn + y_posn
                    if((w = arr[i_k_idx] + arr[k_j_idx]) < arr[i_j_idx]){
                        arr[i_j_idx] = w;
                    }
                }
            }
        }
}

__global__ void gpu_submatrix_fw_krnl_p3(int row_offset, int colm_offset, int *arr, int rnd_id, int blocking_factor, int n){
    int w = 0;
    row_offset = threadIdx.x * blocking_factor;
    for(int b_k = 0; b_k < blocking_factor; b_k++){
            //accessing each element in the submatrix
            for(int b_i = 0; b_i < blocking_factor; b_i++){
                for(int b_j = 0; b_j < blocking_factor; b_j++){
                    //index for i,j
                    int i_j_idx = (row_offset + b_i) * n + (colm_offset * blocking_factor) + b_j; //x_posn + y_posn
                    int i_k_idx = (row_offset + b_i) * n + (rnd_id * blocking_factor) + b_k; //x_posn + y_posn
                    int k_j_idx = (rnd_id * blocking_factor + b_k) * n + (colm_offset * blocking_factor) + b_j; //x_posn + y_posn
                    if((w = arr[i_k_idx] + arr[k_j_idx]) < arr[i_j_idx]){
                        arr[i_j_idx] = w;
                    }
                }  
            }
        }
}

__global__ void gpu_submatrix_fw_krnl_p4(int row_offset, int colm_offset, int *arr, int rnd_id, int blocking_factor, int n){
    int w = 0;
    row_offset = threadIdx.x * blocking_factor;
    colm_offset = blockIdx.x;
    //printf("(x,y) is: (%d, %d)\n", threadIdx.x, blockIdx.x);
    for(int b_k = 0; b_k < blocking_factor; b_k++){
            //accessing each element in the submatrix
            for(int b_i = 0; b_i < blocking_factor; b_i++){
                for(int b_j = 0; b_j < blocking_factor; b_j++){
                    //index for i,j
                    int i_j_idx = (row_offset + b_i) * n + (colm_offset * blocking_factor) + b_j; //x_posn + y_posn
                    int i_k_idx = (row_offset + b_i) * n + (rnd_id * blocking_factor) + b_k; //x_posn + y_posn
                    int k_j_idx = (rnd_id * blocking_factor + b_k) * n + (colm_offset * blocking_factor) + b_j; //x_posn + y_posn
                    if((w = arr[i_k_idx] + arr[k_j_idx]) < arr[i_j_idx]){
                        arr[i_j_idx] = w;
                    }
                }
            }
        }
}


int main(int argc, char** argv) {
    int m, *d;
    // input
    FILE *infile = fopen(argv[1], "r");
    fscanf(infile, "%d %d", &n, &m);
    rounds = ceil(n/(double)blocking_factor);
    d = (int *) malloc(sizeof(int *) * n * n);
    for (int i = 0; i < n * n; ++i) d[i] = INF;
    int a, b, w;
    
    for (int i = 0; i < m; ++i) {
        fscanf(infile, "%d %d %d", &a, &b, &w);
        d[a * n + b] = d[b * n + a] = w;
    }
    fclose(infile);

    
    
    auto start = high_resolution_clock::now();
    //RJ: GPU init
    int *gpu_d;
    hipMalloc((void **)&gpu_d, sizeof(int *) * n * n);
    hipMemcpy(gpu_d, d, sizeof(int)* n * n, hipMemcpyHostToDevice);
    dim3 block_dim(blocking_factor,blocking_factor);
    //RJ: notes
    // N/B rounds --> 3 phases --> B iterations
    // seems like phase2, phase 3 can be parallelized.
    

    printf("Rounds are: %d\n", rounds);
    for (int rnd_id = 0; rnd_id < rounds; rnd_id++){ // number of rounds
        //do 3 phases in each round
        //Phase1 -- computing on the pivot block
        //(1) find which cell to work on. (2) apply blocked FW

        int cell_rowid = rnd_id * blocking_factor; // row_id
        int cell_colmid = rnd_id * blocking_factor; // row_id
        //GPU declaration and kernel launch:
        dim3 dimGrid = (1);
        dim3 dimBlock = 1;
        //start = high_resolution_clock::now();
        gpu_submatrix_fw_krnl_p1<<<dimGrid, dimBlock>>>(cell_rowid, rnd_id, gpu_d, rnd_id, blocking_factor, n);
        hipDeviceSynchronize();


        //Phase2 -- apply fw on the pivot-row and pivot-colm
        // (1) cover row
        // (2) cover colm
        //pivot block is (rnd_id, rnd_id)
        //par1: cover row
        
        dimGrid = (1);
        dimBlock = rounds;

        //start = high_resolution_clock::now();
        gpu_submatrix_fw_krnl_p2<<<dimGrid, dimBlock>>>(cell_rowid, 0, gpu_d, rnd_id, blocking_factor, n);

        //par2: cover colm
        gpu_submatrix_fw_krnl_p3<<<dimGrid, dimBlock>>>(0*blocking_factor, rnd_id, gpu_d, rnd_id, blocking_factor, n);
        hipDeviceSynchronize();
        

        //Phase3 -- apply fw on the non-pivot blocks
        dimGrid = (rounds);
        dimBlock = (rounds);
        gpu_submatrix_fw_krnl_p4<<<dimGrid, dimBlock>>>(0*blocking_factor, 0, gpu_d, rnd_id, blocking_factor, n);
        hipDeviceSynchronize();        
    }
    
    //Obtain back the data from GPU
    hipMemcpy(d, gpu_d, sizeof(int)* n * n, hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    auto stop = high_resolution_clock::now();
    auto duration = duration_cast<microseconds>(stop - start);
    std::cout << ">>>> Time to compute on GPU: " << duration.count()/(1000000.0) << std::endl;
    
    // ouput
    FILE *outfile = fopen(argv[2], "w");
    for (int i = 0; i < n; ++i) {
        for (int j = 0; j < n; ++j) {
            fprintf(outfile, "%d%s",
                (i == j ? 0 : d[i * n + j]),
                (j == n - 1 ? " \n" : " ")
            );
        }
    }
    free(d);
}