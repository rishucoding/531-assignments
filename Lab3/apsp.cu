#include "hip/hip_runtime.h"
#include <chrono>
#include <cstdio>
#include <cstdlib>
#include <fstream>
#include <hip/hip_runtime.h>

#define check(err) __check(err, __LINE__)
void __check(hipError_t err, int line) {
    if (err) {
        fprintf(stderr, "%d: %s\n", line, hipGetErrorString(err));
    }
}

int main(int argc, char** argv) {
    // TODO: Read input from argv[1]
    // TODO: Blocked Floyd-Warshall
    int B = atoi(argv[3]);
    // int round = ceil(n / B);
    for (int r = 0; r < round; r++) {
        // phase 1
        // phase 2
        // phase 3
    }
    // TODO: Write output to argv[2]
}
